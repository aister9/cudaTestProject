#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "rmg.h"
#include <direct.h>
#include "ArrayQueue.h"
#include "DS_timer.h"

#define LAST_FILE_NUMBER 10
#define MATRIX_W 1024
#define MATRIX_H 1024
#define QUEUE_CAPACITY 6

#define PREPROCESS true
#define NORMALCASE true
#define SAVEFORMAT 1 //0 is File, 1 is Vector

__global__ void addSample(int *inputData,int *gpuMap,int *outputData, int w, int h);

int main() {
	srand((unsigned int)time(NULL));

#if PREPROCESS == true
	if (mkdir("matrixs") == -1) {
		printf("matrixs: Folder is already exist..\n");
	}
	if (mkdir("output") == -1) {
		printf("output: Folder is already exist..\n");
	}
	if (mkdir("case2-output") == -1) {
		printf("case2-output: Folder is already exist..\n");
	}

	cout << "Random Matrix Generation ... ";
	for (int i = 0; i < LAST_FILE_NUMBER; i++) {
		string filename = "matrixs\\matrix " + to_string(i) + ".txt";
		random_matrix_generator(filename, MATRIX_W, MATRIX_H);
		int persentage = (double)i / (double)10 * 100;
		if (persentage % 25 == 0) cout << persentage << "% ... ";
	}
	cout << "Complete" << endl;
#endif

	int *gpuMap; int *inputData[QUEUE_CAPACITY]; int *outputData[QUEUE_CAPACITY];
	for (int i = 0; i < QUEUE_CAPACITY; i++) {
		hipMalloc(&inputData[i], sizeof(int) * MATRIX_W * MATRIX_H);
		hipMalloc(&outputData[i], sizeof(int) * MATRIX_W * MATRIX_H);
	}
	hipMalloc(&gpuMap, sizeof(int) * MATRIX_W * MATRIX_H);
	int *cpuMap = new int[MATRIX_W * MATRIX_H];

	for (int i = 0; i < MATRIX_W * MATRIX_H; i++) {
		cpuMap[i] = rand() % 10;
	}
	record_matrix_in_file("cpumap.txt", cpuMap, MATRIX_W, MATRIX_H);

	//gpuMap Generate
	hipMemcpy(gpuMap, cpuMap, sizeof(int) * MATRIX_W * MATRIX_H, hipMemcpyHostToDevice);
	delete[] cpuMap;

	hipStream_t streams[QUEUE_CAPACITY];
	hipEvent_t isEnd[LAST_FILE_NUMBER];
	for (int i = 0; i < QUEUE_CAPACITY; i++) {
		hipStreamCreate(&streams[i]);
	}
	for (int i = 0; i < LAST_FILE_NUMBER; i++)
		hipEventCreate(&isEnd[i]);
	ArrayQueue aq; //input Queue
	ArrayQueue rq; //output Queue
	aq.resize(QUEUE_CAPACITY);
	rq.resize(LAST_FILE_NUMBER);
	//Queue_capacity == 6, MATRIX_W * MATRIX_H * 4 = 31MB * 6 = 186MB
	//aq + rq = 186+310 = 496MB(2.42% use ram : total ram 20gb)
	aq.set_array_size_pinned(MATRIX_W, MATRIX_H);
	rq.set_array_size_pinned(MATRIX_W, MATRIX_H);

	int ind = 0;
	bool error = false;

	DS_timer caseTimer(2);
	caseTimer.initTimers();
	caseTimer.setTimerName(0, "Use Concurrent Kernel(Heterogeneous parallel)");
	caseTimer.setTimerName(1, "Use Normal Case(1 Kernel "+to_string(LAST_FILE_NUMBER)+" LOOP)");
	vector<int *> case1_Vector;
	vector<int *> case2_Vector;

	//////////////////////////////////////////////////////////////////////////
	/*Concurrent Kernel*/
	caseTimer.onTimer(0);
#pragma omp parallel sections
	{
#pragma omp section // file read
		{
			while (true) {
				
				if (aq.isFull()) { //�������¸� ���ѷ���
					continue;
				}
				//�ƴϸ�
				string fname = "matrixs\\matrix " + to_string(ind) + ".txt";
				aq.tailAdder(); //���� ť�� ��ġ�� tail�� �̵�
				if (!read_matrix_in_file(fname, aq.data[aq.tail], MATRIX_W, MATRIX_H)) {
					cout << "error options" << endl;
					error = true;
					break;
				}
				aq.chk=aq.tail;
				ind++;

				if (ind == LAST_FILE_NUMBER) {
					break;
				}
			}
		}
#pragma omp section // run kernel
		{
			int streamInd = 0;
			int fileInd = 0;
			while (true) {
				if (aq.isEmpty()) {
					if (ind == LAST_FILE_NUMBER) break;
					else continue;
				}
				if (error) break;
				if (aq.chk != aq.tail) continue;
				
				hipMemcpyAsync(inputData[streamInd], aq.dequeue(), sizeof(int)*MATRIX_W*MATRIX_H, hipMemcpyHostToDevice, streams[streamInd]);
				addSample << <1, 512, 0, streams[streamInd] >> > (inputData[streamInd], gpuMap, outputData[streamInd], MATRIX_W, MATRIX_H);
				while (rq.isFull()) {
					continue;
				}
				rq.tailAdder();
				hipMemcpyAsync(rq.getTailData(), outputData[streamInd], sizeof(int)*MATRIX_W*MATRIX_H, hipMemcpyDeviceToHost, streams[streamInd]);
				hipEventRecord(isEnd[fileInd++], streams[streamInd]);

				streamInd = (streamInd + 1) % QUEUE_CAPACITY;
			}
		}
#pragma omp section //file output
		{
			int resultInd = 0;
			while (true) {
				if (rq.isEmpty()) {
					if (ind == LAST_FILE_NUMBER) break;
					else continue;
				}
				if (hipEventQuery(isEnd[resultInd]) != hipSuccess) continue;
				
#if SAVEFORMAT == 0
				string filename = "output\\result " + to_string(resultInd) + ".txt";
				if (!record_matrix_in_file(filename, rq.dequeue(), MATRIX_W, MATRIX_H)) {
					cout << "error options : can`t record matrix" << endl;
					error = true;
					break;
				}
#elif SAVEFORMAT == 1
				case1_Vector.push_back(rq.dequeue());
#endif	
				resultInd++;
			}
		}
	}
	caseTimer.offTimer(0);
	//////////////////////////////////////////////////////////////////////////
	
	cout << "--------------------------------------------------------" << endl;
	cout << "CASE 1 END" << endl;
	cout << "--------------------------------------------------------" << endl;

	//////////////////////////////////////////////////////////////////////////
	/*NORMAL CASE*/
	caseTimer.onTimer(1);
#if NORMALCASE == true
	for (int i = 0; i < LAST_FILE_NUMBER; i++) {
		string openfname = "matrixs\\matrix " + to_string(i) + ".txt";
		string outfname = "case2-output\\result " + to_string(i) + ".txt";
		int *inputMatrix = new int[MATRIX_W*MATRIX_H];
		int *outputMatrix = new int[MATRIX_W*MATRIX_H];
		if (!read_matrix_in_file(openfname, inputMatrix, MATRIX_W, MATRIX_H)) {
			cout << "error options" << endl;
			break;
		}
		hipMemcpy(inputData[0], inputMatrix, sizeof(int)*MATRIX_W*MATRIX_H, hipMemcpyHostToDevice);
		addSample << <1, 512 >> > (inputData[0], gpuMap, outputData[0], MATRIX_W, MATRIX_H);
		hipDeviceSynchronize();
		hipMemcpy(outputMatrix, outputData[0], sizeof(int)*MATRIX_W*MATRIX_H, hipMemcpyDeviceToHost);

#if SAVEFORMAT == 0
		if (!record_matrix_in_file(outfname, outputMatrix, MATRIX_W, MATRIX_H)) {
			cout << "error options : can`t record matrix" << endl;
			error = true;
			break;
		}
#elif SAVEFORMAT == 1
		case2_Vector.push_back(outputMatrix);
#endif	
		
	}
#endif
	caseTimer.offTimer(1);
	//////////////////////////////////////////////////////////////////////////

	cout << "--------------------------------------------------------" << endl;
	cout << "CASE 2 END" << endl;
	cout << "--------------------------------------------------------" << endl;

	caseTimer.printTimer();

	ofstream timerlog;
	timerlog.open("Timer Log.txt", ios::app);

#if SAVEFORMAT == 0
	timerlog << "MODE : OUTPUT FILE" << endl;
#elif SAVEFORMAT == 1
	timerlog << "MODE : OUTPUT VECTOR" << endl;
#endif	
	timerlog << "MATRIX SIZE : " << MATRIX_W << "X" << MATRIX_H << " - LOOP COUNT : " << LAST_FILE_NUMBER << endl;
	timerlog.close();

	caseTimer.printToFile("Timer Log.txt");

	for (int i = 0; i < QUEUE_CAPACITY; i++) {
		hipStreamDestroy(streams[i]);
		hipEventDestroy(isEnd[i]);
	}
	for (int i = 0; i < QUEUE_CAPACITY; i++) {
		hipFree(inputData[i]);
		hipFree(outputData[i]);
	}
	aq.cudaFreeAllMembers();
	hipFree(gpuMap);
}

__global__ void addSample(int *inputData, int *gpuMap, int *outputData, int w, int h) {
	int tIdx = threadIdx.x;
	int totalThreads = blockDim.x;

	for (int i = tIdx; i < w*h; i += totalThreads) {
		outputData[i] = inputData[i] + gpuMap[i];
	}

	return;
}


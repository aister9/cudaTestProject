#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "rmg.h"
#include <direct.h>
#include "ArrayQueue.h"

#define LAST_FILE_NUMBER 10
#define MATRIX_W 3840
#define MATRIX_H 2160
#define QUEUE_CAPACITY 6

#define PREPROCESS false

__global__ void addSample(int *inputData,int *gpuMap,int *outputData, int w, int h);

int main() {
	srand((unsigned int)time(NULL));

#if PREPROCESS == true
	if (mkdir("matrixs") == -1) {
		printf("Folder is already exist..\n");
	}

	cout << "Random Matrix Generation ... ";
	for (int i = 0; i < LAST_FILE_NUMBER; i++) {
		string filename = "matrixs\\matrix " + to_string(i) + ".txt";
		random_matrix_generator(filename, MATRIX_W, MATRIX_H);
		int persentage = (double)i / (double)10 * 100;
		if (persentage % 25 == 0) cout << persentage << "% ... ";
	}
	cout << "Complete" << endl;
#endif

	int *gpuMap; int *inputData[QUEUE_CAPACITY]; int *outputData[QUEUE_CAPACITY];
	for (int i = 0; i < QUEUE_CAPACITY; i++) {
		hipMalloc(&inputData[i], sizeof(int) * MATRIX_W * MATRIX_H);
		hipMalloc(&outputData[i], sizeof(int) * MATRIX_W * MATRIX_H);
	}
	hipMalloc(&gpuMap, sizeof(int) * MATRIX_W * MATRIX_H);
	int *cpuMap = new int[MATRIX_W * MATRIX_H];

	for (int i = 0; i < MATRIX_W * MATRIX_H; i++) {
		cpuMap[i] = rand() % 10;
	}

	//gpuMap Generate
	hipMemcpy(gpuMap, cpuMap, sizeof(int) * MATRIX_W * MATRIX_H, hipMemcpyHostToDevice);
	delete[] cpuMap;

	hipStream_t streams[QUEUE_CAPACITY];
	for (int i = 0; i < QUEUE_CAPACITY; i++) {
		hipStreamCreate(&streams[i]);
	}
	ArrayQueue aq; //input Queue
	ArrayQueue rq; //output Queue
	aq.resize(QUEUE_CAPACITY);
	rq.resize(LAST_FILE_NUMBER);
	//Queue_capacity == 6, MATRIX_W * MATRIX_H * 4 = 31MB * 6 = 186MB
	//aq + rq = 186+310 = 496MB(2.42% use ram : total ram 20gb)
	aq.set_array_size_pinned(MATRIX_W, MATRIX_H);
	rq.set_array_size_pinned(MATRIX_W, MATRIX_H);

	int ind = 0;
	bool error = false;
#pragma omp parallel sections
	{
#pragma omp section // file read
		{
			while (true) {
				
				if (aq.isFull()) { //�������¸� ���ѷ���
					continue;
				}
				//�ƴϸ�
				string fname = "matrixs\\matrix " + to_string(ind) + ".txt";;
				aq.tailAdder(); //���� ť�� ��ġ�� tail�� �̵�
				if (!read_matrix_in_file(fname, aq.getTailData(), MATRIX_W, MATRIX_H)) {
					cout << "error options" << endl;
					error = true;
					break;
				}

				ind++;

				if (ind == LAST_FILE_NUMBER) {
					break;
				}
			}
		}
#pragma omp section // run kernel
		{
			int streamInd = 0;
			while (true) {
				if (aq.isEmpty()) {
					if (ind == LAST_FILE_NUMBER) break;
					else continue;
				}
				if (error) break;

				hipMemcpyAsync(inputData[streamInd], aq.dequeue(), sizeof(int)*MATRIX_W*MATRIX_H, hipMemcpyHostToDevice, streams[streamInd]);
				addSample << <1, 512, 0, streams[streamInd] >> > (inputData[streamInd], gpuMap, outputData[streamInd], MATRIX_W, MATRIX_H);
				
				streamInd = (streamInd + 1) % QUEUE_CAPACITY;
			}
		}
	}
	for (int i = 0; i < QUEUE_CAPACITY; i++) {
		hipStreamDestroy(streams[i]);
	}
	for (int i = 0; i < QUEUE_CAPACITY; i++) {
		hipFree(inputData[i]);
		hipFree(outputData[i]);
	}
	aq.cudaFreeAllMembers();
	hipFree(gpuMap);
}

__global__ void addSample(int *inputData, int *gpuMap, int *outputData, int w, int h) {
	int tIdx = threadIdx.x;
	int totalThreads = blockDim.x;

	for (int i = tIdx; i < w*h; i += totalThreads) {
		outputData[i] = inputData[i] + gpuMap[i];
	}

	return;
}

